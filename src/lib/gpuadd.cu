#include "hip/hip_runtime.h"
/************************************************************************
 Sample CUDA MEX kernel code written by Fang Liu (leoliuf@gmail.com).
************************************************************************/

#ifndef _GPU_ADD_CU_
#define _GPU_ADD_CU_

#include <stdio.h>

#include "gpuadd.cuh"

__global__ void gpuaddkernel(double *d_A, double *d_B, double *d_C, size_t Am, size_t An)
{
    /* index */
	unsigned int tid = blockIdx.x * blockDim.y + threadIdx.y; /* thread id in matrix*/
	/* strip */
	unsigned int strip = gridDim.x * blockDim.y;

	while (1) {
	 if (tid  < Am * An){
		d_C[tid] = d_A[tid] + d_B[tid];
	 }
	 else{
	    break;
	 }
	 tid += strip;
	}
}

void gpuadd(double *A, double *B, double *C, size_t Am, size_t An)
{

  /* set GPU grid & block configuration */
  hipDeviceProp_t deviceProp;
  memset( &deviceProp, 0, sizeof(deviceProp));
  if( hipSuccess != hipGetDeviceProperties(&deviceProp,0)){
    printf( "\n%s", hipGetErrorString(hipGetLastError()));
    return;
  }

  dim3 dimGridImg(8,1,1);
  dim3 dimBlockImg(1,64,1);
	
  /* allocate device memory for matrices */
  double *d_A = NULL;
  hipMalloc( (void**) &d_A, Am * An * sizeof(double)) ;
	hipMemcpy( d_A, A, Am * An * sizeof(double), hipMemcpyHostToDevice) ;
  double *d_B = NULL;
  hipMalloc( (void**) &d_B, Am * An * sizeof(double)) ;
	hipMemcpy( d_B, B, Am * An * sizeof(double), hipMemcpyHostToDevice) ;
  double *d_C = NULL;
  hipMalloc( (void**) &d_C, Am * An * sizeof(double)) ;
    
	/* call GPU kernel for addition */
	gpuaddkernel<<< dimGridImg, dimBlockImg >>>(d_A, d_B, d_C, Am, An);
	hipDeviceSynchronize();
    
  /* copy result from device */
	hipMemcpy( C, d_C, Am * An * sizeof(double), hipMemcpyDeviceToHost) ;


  /* free GPU memory */
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

}

#endif
