#include "hip/hip_runtime.h"
/************************************************************************
 Sample CUDA MEX kernel code written by Fang Liu (leoliuf@gmail.com).
************************************************************************/

#ifndef _GPU_ADD_CU_
#define _GPU_ADD_CU_

#include <stdio.h>

#include "gpuadd.cuh"

__global__ void gpuaddkernel(double *d_A, double *d_B, double *d_C, size_t Am, size_t An)
{
  int x = threadIdx.x + blockDim.x * blockIdx.x;
  int y = threadIdx.y + blockDim.y * blockIdx.y;
  int ind = x + An*y; //derive linear index
  if (x<An && y<Am) d_C[ind] = d_A[ind] + d_B[ind];
}

void gpuadd(double *A, double *B, double *C, size_t Am, size_t An)
{

  /* set GPU grid & block configuration */
  hipDeviceProp_t deviceProp;
  memset( &deviceProp, 0, sizeof(deviceProp));
  if( hipSuccess != hipGetDeviceProperties(&deviceProp,0)){
    printf( "\n%s", hipGetErrorString(hipGetLastError()));
    return;
  }

  dim3 block = dim3(32,8,1); // 32*8*1 = 256 threads per block
  // ensure enough blocks to cover w * h elements (round up)
  dim3 grid = dim3( ( An + block.x -1 ) / block.x, ( Am + block.y - 1 ) / block.y, 1);
	
  /* allocate device memory for matrices */
  double *d_A = NULL;
  hipMalloc( (void**) &d_A, Am * An * sizeof(double)) ;
	hipMemcpy( d_A, A, Am * An * sizeof(double), hipMemcpyHostToDevice) ;
  double *d_B = NULL;
  hipMalloc( (void**) &d_B, Am * An * sizeof(double)) ;
	hipMemcpy( d_B, B, Am * An * sizeof(double), hipMemcpyHostToDevice) ;
  double *d_C = NULL;
  hipMalloc( (void**) &d_C, Am * An * sizeof(double)) ;
    
	/* call GPU kernel for addition */
	gpuaddkernel<<< grid, block >>>(d_A, d_B, d_C, Am, An);
	hipDeviceSynchronize();
    
  /* copy result from device */
	hipMemcpy( C, d_C, Am * An * sizeof(double), hipMemcpyDeviceToHost) ;


  /* free GPU memory */
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

}

#endif
