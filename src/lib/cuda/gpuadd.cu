#include "hip/hip_runtime.h"
#ifndef _GPU_ADD_CU_
#define _GPU_ADD_CU_

#include <stdio.h> //size_t
#include <math.h> //fabs
#include <cstring> //memcpy
#include <iostream> //cout

//CUDA libraries
#include "hipblas.h"

//HEADER file
#include "gpuadd.cuh"

//the array index of a matrix element in row “r” and column “c” can be computed via the following macro
#define IDX2C(r,c,rows) (((c)*(rows))+(r))

__global__ void gpuaddkernel(double *d_A, double *d_B, double *d_C, size_t Am, size_t An)
{
  int x = threadIdx.x + blockDim.x * blockIdx.x;
  int y = threadIdx.y + blockDim.y * blockIdx.y;
  int ind = x + An*y; //derive linear index
  if (x<An && y<Am) d_C[ind] = d_A[ind] + d_B[ind];
}

void gpuaddcublas(double *A, double *B, double *C, size_t Am, size_t An)
{

    hipError_t cudaStat;    
    hipblasStatus_t stat;
    hipblasHandle_t handle;

    double *d_A, *d_C;
    if (!A || !B || !C) {
        printf ("Error in gpuaddcublas: input failed\n");
        return;
    }
    
    cudaStat = hipMalloc ((void**)&d_A, Am*An*sizeof(double));
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed\n");
        return;
    }
    cudaStat = hipMalloc ((void**)&d_C, Am*An*sizeof(double));
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed\n");
        return;
    }

    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return;
    }

    //for (size_t ii=0; ii<Am*An; ii++) printf("A=%f\n",A[ii]); 

    stat = hipblasSetVector(Am*An, sizeof(double), (void*)A, 1, (void*)d_A, 1);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data download for A failed: %d\n", stat);
        hipFree (d_A);
        hipblasDestroy(handle);
        return;
    }
    stat = hipblasSetVector(Am*An, sizeof(double), (void*)B, 1, (void*)d_C, 1);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data download for B failed\n");
        hipFree (d_A);
        hipFree (d_C);
        hipblasDestroy(handle);
        return;
    }

  //add two matrices (represented as vectors)
    double alpha = 1.;
    stat =  hipblasAxpyEx (handle,Am*An, &alpha, HIP_R_64F, (void *)d_A, HIP_R_64F, 1, (void *)d_C, HIP_R_64F,  1, HIP_R_64F);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data adding failed\n");
        hipFree (d_A);
        hipFree (d_C);
        hipblasDestroy(handle);
        return;
    }

    stat = hipblasGetVector(Am*An, sizeof(double), (void *)d_C, 1, (void*)C, 1);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data upload failed\n");
        hipFree (d_A);
        hipFree (d_C);
        hipblasDestroy(handle);
        return;
    }
   hipFree (d_A);
   hipFree (d_C);
   hipblasDestroy(handle);
   return;

}

void gpuadd(double *A, double *B, double *C, size_t Am, size_t An)
{

  /* set GPU grid & block configuration */
  hipDeviceProp_t deviceProp;
  memset( &deviceProp, 0, sizeof(deviceProp));
  if( hipSuccess != hipGetDeviceProperties(&deviceProp,0)){
    printf( "\n%s", hipGetErrorString(hipGetLastError()));
    return;
  }

  dim3 block = dim3(32,8,1); // 32*8*1 = 256 threads per block
  // ensure enough blocks to cover w * h elements (round up)
  dim3 grid = dim3( ( An + block.x -1 ) / block.x, ( Am + block.y - 1 ) / block.y, 1);


//First add two pointers with specified kernel

  /* allocate device memory for matrices */
  double *d_A = NULL;
  hipMalloc( (void**) &d_A, Am * An * sizeof(double)) ;
	hipMemcpy( d_A, A, Am * An * sizeof(double), hipMemcpyHostToDevice) ;
  double *d_B = NULL;
  hipMalloc( (void**) &d_B, Am * An * sizeof(double)) ;
	hipMemcpy( d_B, B, Am * An * sizeof(double), hipMemcpyHostToDevice) ;
  double *d_C = NULL;
  hipMalloc( (void**) &d_C, Am * An * sizeof(double)) ;
    
  std::cout << "Values/Vectors/Matrices are added using a self-implemented kernel" << std::endl;
	/* call GPU kernel for addition */
	gpuaddkernel<<< grid, block >>>(d_A, d_B, d_C, Am, An);
	hipDeviceSynchronize();
    
  /* copy result from device */
  double *C_kernel = new double [Am*An];
	hipMemcpy( C_kernel, d_C, Am * An * sizeof(double), hipMemcpyDeviceToHost) ;


  /* free GPU memory */
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

//Now add pointers with cublas
  std::cout << "Addtionally, the same values are added using cublas" << std::endl;
  double *C_cublas = new double [Am*An];
	gpuaddcublas(A, B, C_cublas, Am, An);

//Compare results
  std::cout << "And now both results will be compared..." << std::endl;;
bool equal = true;
for (size_t ii=0; ii<Am*An; ii++)
  if (!(fabs(C_kernel[ii]-C_cublas[ii])<0.001))
    equal = false;

if (equal)
{
  std::cout << "Yeah, both arrays have the same values" << std::endl << std::endl;
  std::memcpy( (void*)C, (void*) C_kernel, Am * An * sizeof(double) );
  free(C_kernel);
  free(C_cublas);
}
else
{
  std::cout << "Oh no, cublas and your own kernel differ too much in results." << std::endl;
  std::cout << "Copy kernel results now, but handle with caution." << std::endl << std::endl;
  std::memcpy( (void*)C, (void*) C_kernel, Am * An * sizeof(double) );
  free(C_kernel);
  free(C_cublas);
}

}

#endif
